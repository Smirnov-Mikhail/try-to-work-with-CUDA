#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cuProc.h"
#define BLOCKSIZE 16
#define GridSize(size) (size/BLOCKSIZE + 1)

__global__ void addContours(int* out_gray, int* out_alpha, int* graylvl, int* alpha, /* from original image */
	int height, int width,  /* size of original image */
	int deviation /* the width of liminiscence strip */
) {
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if (alpha[x + y * width] == 0 /* if pixel is transparent */) {
		int mindev = 2 * deviation;
		for (int i = -deviation; i <= deviation; i++) {
			for (int j = -deviation; j <= deviation; j++) {
				if (x + i >= 0 && x + i < width && y + j >= 0 && y + j < height) {
					if (graylvl[x + i + (y + j) * width] < 128 && alpha[x + i + (y + j) * width] != 0 && mindev > min(abs(i), abs(j))) {
						mindev = min(abs(i), abs(j));
					}
				}
			}
		}
		if (mindev < 2 * deviation) {
			out_alpha[x + y * width] = 255 / mindev;
			out_gray[x + y * width] = 255;
		}
	}
	else {
		out_alpha[x + y * width] = alpha[x + y * width];
		out_gray[x + y * width] = graylvl[x + y * width];
	}
}

void process(int* img, int* alpha, int height, int width, int dev) {
	//	hipSetDevice(0);
	hipError_t err;
	int* img_dev;
	int* out_img_dev;
	int* alpha_dev;
	int* out_alpha_dev;
	int alloc_size = height*width * sizeof(int);

	err = hipMalloc((void**)&img_dev, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
	err = hipMalloc((void**)&alpha_dev, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
	err = hipMalloc((void**)&out_img_dev, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
	err = hipMalloc((void**)&out_alpha_dev, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}

	err = hipMemcpy(img_dev, img, alloc_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to copy h2d!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
	err = hipMemcpy(alpha_dev, alpha, alloc_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("ERROR: unable to copy h2d!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}

	addContours << < dim3(width / BLOCKSIZE, height / BLOCKSIZE), dim3(BLOCKSIZE, BLOCKSIZE) >> >(out_img_dev, out_alpha_dev, img_dev, alpha_dev, height, width, dev);
	hipDeviceSynchronize();
	err = hipMemcpy(img, out_img_dev, alloc_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("ERROR: unable to copy d2h!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
	err = hipMemcpy(alpha, out_alpha_dev, alloc_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("ERROR: unable to copy d2h!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}
	hipFree(alpha_dev);
	hipFree(img_dev);
	hipFree(out_img_dev);
	hipFree(out_alpha_dev);
	//	hipDeviceReset();
}