#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <ctime>

using namespace std;

#define N (1024*128)
#define M (100000)

__global__ void cudakernel(float *buf)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	buf[i] = 1.0001f * i / N + 0.0002f;
	for (int j = 0; j < M; j++)
		buf[i] = buf[i] * buf[i] - 0.25f;
	buf[i] += (float)i / (float)N;
}

int main()
{
	float data[N];
	float *d_data;
	unsigned int start_time = clock();
	hipMalloc(&d_data, N * sizeof(float));
	cudakernel << <N / 256, 256 >> >(d_data);
	hipDeviceSynchronize();
	hipMemcpy(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_data);
	unsigned int end_time = clock(); // �������� �����
	unsigned int search_time = end_time - start_time; // ������� �����
	cout << "runtime = " << search_time / 1000.0 << endl;
	while (true)
	{
		int index;
		printf("Enter an index: ");
		cin >> index;
		printf("data[%d] = %f\n", index, data[index]);
	}
}