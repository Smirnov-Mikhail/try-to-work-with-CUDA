#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void addKernel(int *c, int *a, int *b) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];		// �������� ���� �������� �� ���
}
int main() {
	const int N = 100;
	int *a_dev; int *b_dev; int *c_dev;	// ���������� ���������� (������ ��� ��� � ���)

	hipMallocManaged(&a_dev, N * sizeof(int));		// ��������� Unified Memory ��� ��������
	hipMallocManaged(&b_dev, N * sizeof(int));
	hipMallocManaged(&c_dev, N * sizeof(int));
	hipDeviceSynchronize();
	for (int i = 0; i < N; i++) { a_dev[i] = 1; b_dev[i] = 2; }	// ������������ �������� �� ���

	addKernel << <1, N >> >(c_dev, a_dev, b_dev);	// ����� ������� �������� �� ���
	hipDeviceSynchronize();
	int res = 0;
	for (int i = 0; i < N; i++) { res += c_dev[i]; }		// ������������ ��������������� ������� �� ���
	return 0;
}
