#include "hip/hip_runtime.h"
#define CUDA_API_PER_THREAD_DEFAULT_STREAM
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <cstdlib>
#include <cstdio>
#include <iostream>
#define STRCOUNT 2

__global__ void vecAdd(double *res, double *inA, double *inB, size_t n) {
	int x = blockDim.x * blockIdx.x + threadIdx.x;
	if (x >= n) return;
	res[x] = inA[x] + inB[x];
}

void add() {
	size_t N = 10000;
	double *A = new double[N];
	double *B = new double[N];
	double *C = new double[N];
	double *dev_A;
	double *dev_B;
	double *dev_C;
	hipError_t err;
	int alloc_size = N * sizeof(double);

	hipStream_t workerstreams[STRCOUNT];
	int str_block = N / STRCOUNT;
	err = hipMalloc((void **)&dev_A, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}

	err = hipMalloc((void **)&dev_B, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}

	err = hipMalloc((void **)&dev_C, alloc_size);
	if (err != hipSuccess) {
		printf("ERROR: unable to  allocate!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}

	for (int i = 0; i < N; i++) {
		A[i] = i;
		B[i] = i;
	}

	for (int i = 0; i < STRCOUNT; i++) {
		hipStreamCreate(&workerstreams[i]);
		int k = i * str_block;
		int memsize = str_block * sizeof(double);
		if (i == STRCOUNT - 1) memsize = (N - k) * sizeof(double);

		err = hipMemcpyAsync(&dev_A[k], &A[k], memsize, hipMemcpyHostToDevice, workerstreams[i]);
		if (err != hipSuccess) {
			printf("ERROR: unable to copy h2d!\n");
			std::cerr << "Err is " << hipGetErrorString(err)
				<< std::endl;
		}

		err = hipMemcpyAsync(&dev_B[k], &B[k], memsize, hipMemcpyHostToDevice, workerstreams[i]);
		if (err != hipSuccess) {
			printf("ERROR: unable to copy h2d!\n");
			std::cerr << "Err is " << hipGetErrorString(err)
				<< std::endl;
		}
	}
	for (int i = 0; i < STRCOUNT; i++) {
		int k = i * str_block;
		int memsize = str_block * sizeof(double);
		if (i == STRCOUNT - 1) memsize = (N - k) * sizeof(double);
		vecAdd << <memsize, 1, 0,
			workerstreams[i] >> >(&dev_C[k], &dev_A[k], &dev_B[k], memsize);
	}
	hipDeviceSynchronize();
	err = hipMemcpy(C, dev_C, alloc_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("ERROR: unable to copy h2d!\n");
		std::cerr << "Err is " << hipGetErrorString(err) << std::endl;
	}

	for (int i = 0; i < N; i++) {
		std::cout << A[i] << " + " << B[i] << " = " << C[i]
			<< std::endl;
	}
	hipFree(dev_A);
	hipFree(dev_B);
	hipFree(dev_C);
}

int main() {
	add();
	return 0;
}