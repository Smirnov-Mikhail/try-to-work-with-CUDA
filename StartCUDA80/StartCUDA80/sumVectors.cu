#include "hip/hip_runtime.h"
#include <>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void addKernel(int *c, int *a, int *b) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];		// �������� ���� �������� �� ���
}
int main() {
	const int N = 100; 
	int size = N * sizeof(int);
	int *a_dev; 
	int *b_dev; 
	int *c_dev;	// ���������� ���������� ��� ���

	int *c_host = (int*)malloc(size); 	
	int *a_host = (int*)malloc(size); 	
	int *b_host = (int*)malloc(size); // ��������� ������ ���

	hipMalloc(&a_dev, size); 	
	hipMalloc(&b_dev, size); 	
	hipMalloc(&c_dev, size); // ��������� ������ ���
	hipDeviceSynchronize();
	for (int i = 0; i < N; i++) 
	{ a_host[i] = 1; b_host[i] = 2; } 		// ������������� �������� �� �����

	hipMemcpy(a_dev, a_host, size, hipMemcpyHostToDevice);	// ����������� �������� �� ���
	hipMemcpy(b_dev, b_host, size, hipMemcpyHostToDevice);
	addKernel << <1, N >> >(c_dev, a_dev, b_dev);			// ����� ������� �� ���
	hipDeviceSynchronize();
	hipMemcpy(c_host, c_dev, size, hipMemcpyDeviceToHost);	// ����������� ���������� �� ����
	int res = 0;
	for (int i = 0; i < N; i++) { res += c_host[i]; }			// �������� ���� ��������
	cout << "result: " << res << endl;
	return 0;
}
