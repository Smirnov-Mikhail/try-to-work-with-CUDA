#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>


using namespace std;

void DisplayHeader()
{
	const int kb = 1024;
	const int mb = kb * kb;

	wcout << "CUDA version:   v" << CUDART_VERSION << endl;
	int devCount;
	hipGetDeviceCount(&devCount);

	for (int i = 0; i < devCount; ++i)
	{
		hipDeviceProp_t props;
		hipGetDeviceProperties(&props, i);
		wcout << "Device name: " << props.name << endl;
		wcout << "Compute capability: " << props.major << "." << props.minor << endl;
		wcout << "Global memory:   " << props.totalGlobalMem / mb << "mb" << endl;
		wcout << "Shared memory:   " << props.sharedMemPerBlock / kb << "kb" << endl;
		wcout << "Constant memory: " << props.totalConstMem / kb << "kb" << endl;
		wcout << "Block registers: " << props.regsPerBlock << endl << endl;

		wcout << "Warp size:         " << props.warpSize << endl;
		wcout << "Threads per block: " << props.maxThreadsPerBlock << endl;
		wcout << "Max block dimensions: [ " << props.maxThreadsDim[0] << ", " << props.maxThreadsDim[1] << ", " << props.maxThreadsDim[2] << " ]" << endl;
		wcout << "Max grid dimensions:  [ " << props.maxGridSize[0] << ", " << props.maxGridSize[1] << ", " << props.maxGridSize[2] << " ]" << endl;
		wcout << endl;
	}
}

int main() {
	DisplayHeader();
	return 0;
}